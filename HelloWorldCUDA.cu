#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;
__global__ void helloFromGPU() {
    printf("Hello from GPU!\n");
}

int main() {
    // Launch the kernel with 1 block and 1 thread
    helloFromGPU<<<1, 1>>>();

    // Wait for the GPU to finish
    hipDeviceSynchronize();

    std::cout << "Hello from CPU!" << endl;
    return 0;
}
